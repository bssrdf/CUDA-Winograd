#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <string.h>
#include <stdio.h>
#include <string.h>
#include <float.h>
#include <math.h>
#include <assert.h>
#include <xmmintrin.h>
#include <immintrin.h>

//#include "hipDNN.h"
#include "util.h"
#include "Kernel256_one.h"


#define cudaCheckError() {																\
	hipError_t e=hipGetLastError();													\
	if(e!=hipSuccess) {																\
		printf("Cuda failure %s:%d:'%s'\n",__FILE__,__LINE__,hipGetErrorString(e));	\
		exit(EXIT_FAILURE);																\
	}																					\
}



__global__ void kernel_1024_one_256(float *A, float *B, float *bnBias, float *bnScale, float *C) {
	int tile = blockIdx.x, in_channel = threadIdx.x, line = threadIdx.y;
	int ind = line*256 + in_channel;

	extern __shared__ float shared_[];
	float *weights = shared_ + 1024*4, *output = weights + 256*16, *input = shared_;
	float *bias = output + 4*256, *scale = bias + 256;

	for (int i = 0; i < 4; i++)
		input[ind + i*1024] = A[tile*4096 + i*1024 + ind];
	bias[in_channel] = bnBias[in_channel];
	scale[in_channel] = bnScale[in_channel];
	output[ind] = 0.0f;
	__syncthreads();

	for (int k = 0; k < 1024; k += 16) {
		float *B_start = B + k*256;
		for (int i = 0; i < 4; i++)
			weights[ind + i*1024] = B_start[i*1024 + ind];
		__syncthreads();

		float *A_start = input + k;
		for (int p = 0; p < 16; p++) {
			output[ind] += A_start[line*1024 + p] * weights[in_channel + p*256];
		}
		__syncthreads();
	}

	float *C_start = C + tile*1024, res = scale[in_channel] * output[ind] + bias[in_channel];
	C_start[ind] = res > 0 ? res : 0;
}


int kernel_256_1_in() {
	float *input = get_parameter(inputName256one, 14*14*1024);
	float *weight = get_parameter(weightName256one, 256*1024);

	float *bnBias = get_parameter(bnBiasName256one, 256);
	float *bnScale = get_parameter(bnScaleName256one, 256);
	float *bnBias_myKernel = get_parameter(bnBias_myKernel_Name256one, 256);
	float *bnScale_myKernel = get_parameter(bnScale_myKernel_Name256one, 256);
	float *eMeanName = get_parameter(eMeanName256one, 256);
	float *eVarName = get_parameter(eVarName256one, 256);

	float *input_, *output_, *weight_, *bnBias_, *bnScale_, *eMeanName_, *eVarName_;

	int nInput = 14*14*1024, nOutput = 14*14*256, nWeights = 256*1024;
	float tmp[nOutput], tmp_cudnn[nOutput];

	uint64_t nT1 = 0, nT2 = 0, nT1_cudnn = 0, nT2_cudnn = 0;
	hipError_t s;

	/////////////////////////////////

	// My Kernel

	/////////////////////////////////

	/*  1. Data preparation  */
	hipMalloc((void **) &input_, nInput<<3);
	hipMalloc((void **) &output_, nOutput<<2);
	hipMalloc((void **) &weight_, nWeights<<2);
	hipMalloc((void **) &bnBias_, 256<<2);
	hipMalloc((void **) &bnScale_, 256<<2);

	hipMemcpy(input_, input, nInput<<2, hipMemcpyHostToDevice);
	hipMemcpy(weight_, weight, nWeights<<2, hipMemcpyHostToDevice);
	hipMemcpy(bnBias_, bnBias_myKernel, 256<<2, hipMemcpyHostToDevice);
	hipMemcpy(bnScale_, bnScale_myKernel, 256<<2, hipMemcpyHostToDevice);


	/*  2. Computing  */
	nT1 = getTimeMicroseconds64();

	kernel_1024_one_256 <<<dim3(49), dim3(256, 4), (4*1024 + 16*256 + 4*256 + 2*256)<<2 >>> (input_, weight_, bnBias_, bnScale_, output_);

	//cudaCheckError();
	hipDeviceSynchronize();

	nT2 = getTimeMicroseconds64();
	printf("TotalTime = %d us\n", nT2-nT1);


	/*  3. Copy back and free  */
	s = hipMemcpy(tmp, output_, nOutput<<2, hipMemcpyDeviceToHost);
	printf("%s\n", hipGetErrorName(s));
	cudaCheckError();

	free(bnBias_myKernel);
	free(bnScale_myKernel);

	
//	output_checker(tmp, tmp_cudnn, 14, 256, 0);

	return ((nT2-nT1) << 16);
}



__global__ void kernel_256_one_1024(float *A, float *B, float *bnBias, float *bnScale, float *C) {
	int tile = blockIdx.x, part = blockIdx.y, in_channel = threadIdx.x, line = threadIdx.y;
	int ind = line*256 + in_channel;

	extern __shared__ float shared_[];
	float *weights = shared_ + 256*4, *output = weights + 256*32, *input = shared_;
	float *bias = output + 4*256, *scale = bias + 256;

	input[ind] = A[tile * 1024 + ind];
	bias[in_channel] = bnBias[part*256 + in_channel];
	scale[in_channel] = bnScale[part*256+ in_channel];
	output[ind] = 0.0f;
	__syncthreads();

	for (int k = 0; k < 256; k += 32) {
		for (int i = 0; i < 8; i++)
			weights[ind + 1024*i] = B[(k + i*4 + line)*1024 + part*256 + in_channel];
		__syncthreads();

		float *A_start = input + k;
		for (int p = 0; p < 32; p++) {
			output[ind] += A_start[line*256 + p] * weights[in_channel + p*256];
		}
		__syncthreads();
	}

	float *C_start = C + tile*4096 + part*256;
	C_start[line * 1024 + in_channel] = scale[in_channel] * output[ind] + bias[in_channel];
}


int kernel_256_1_out() {
	float *input = get_parameter(inputName256one, 14*14*256);
	float *weight = get_parameter(weightName256one, 256*1024);

	float *bnBias = get_parameter(bnBiasName256one, 1024);
	float *bnScale = get_parameter(bnScaleName256one, 1024);
	float *bnBias_myKernel = get_parameter(bnBias_myKernel_Name256one, 1024);
	float *bnScale_myKernel = get_parameter(bnScale_myKernel_Name256one, 1024);
	float *eMeanName = get_parameter(eMeanName256one, 1024);
	float *eVarName = get_parameter(eVarName256one, 1024);

	float *input_, *output_, *weight_, *bnBias_, *bnScale_, *eMeanName_, *eVarName_;

	int nInput = 14*14*256, nOutput = 14*14*1024, nWeights = 256*1024;
	float tmp[nOutput], tmp_cudnn[nOutput];

	uint64_t nT1 = 0, nT2 = 0, nT1_cudnn = 0, nT2_cudnn = 0;
	hipError_t s;

	/////////////////////////////////

	// My Kernel

	/////////////////////////////////

	/*  1. Data preparation  */
	hipMalloc((void **) &input_, nInput<<3);
	hipMalloc((void **) &output_, nOutput<<2);
	hipMalloc((void **) &weight_, nWeights<<2);
	hipMalloc((void **) &bnBias_, 1024<<2);
	hipMalloc((void **) &bnScale_, 1024<<2);

	hipMemcpy(input_, input, nInput<<2, hipMemcpyHostToDevice);
	hipMemcpy(weight_, weight, nWeights<<2, hipMemcpyHostToDevice);
	hipMemcpy(bnBias_, bnBias_myKernel, 1024<<2, hipMemcpyHostToDevice);
	hipMemcpy(bnScale_, bnScale_myKernel, 1024<<2, hipMemcpyHostToDevice);


	/*  2. Computing  */
	nT1 = getTimeMicroseconds64();

	kernel_256_one_1024 <<<dim3(49, 4), dim3(256, 4), (4*256 + 32*256 + 4*256 + 2*256)<<2 >>> (input_, weight_, bnBias_, bnScale_, output_);

	cudaCheckError();
	hipDeviceSynchronize();

	nT2 = getTimeMicroseconds64();
	printf("TotalTime = %d us\n", nT2-nT1);


	/*  3. Copy back and free  */
	s = hipMemcpy(tmp, output_, nOutput<<2, hipMemcpyDeviceToHost);
	printf("%s\n", hipGetErrorName(s));
	cudaCheckError();

	free(bnBias_myKernel);
	free(bnScale_myKernel);


//	output_checker(tmp, tmp_cudnn, 14, 1024, 0);

	return ((nT2-nT1) << 16);
}
