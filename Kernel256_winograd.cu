#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <string.h>
#include <stdio.h>
#include <string.h>
#include <float.h>
#include <math.h>
#include <assert.h>
#include <xmmintrin.h>
#include <immintrin.h>

#include "hipDNN.h"
#include "util.h"
#include "Kernel256_winograd.h"


#define cudaCheckError() {																\
	hipError_t e=hipGetLastError();													\
	if(e!=hipSuccess) {																\
		printf("Cuda failure %s:%d:'%s'\n",__FILE__,__LINE__,hipGetErrorString(e));	\
		exit(EXIT_FAILURE);																\
	}																					\
}

#define MY_KERNEL 0

#define d(input, i, j, Inz) ( input[Inz + i*768 + (j<<7)] )
__global__ void kernel_256_winograd_BtdB(float *pInputs, float *pOutputs) {
	int Inx = blockIdx.x<<2, Iny0 = blockIdx.y<<2, Part = blockIdx.z, Iny1 = threadIdx.y, Inz = threadIdx.x;
	int Iny = Iny0+Iny1, stride_r = 4096, stride_c = 256; // 4096 = 16*256
	int c_glb_start = Inx*stride_r + Iny*stride_c + Inz + (Part<<7), c_input = Iny1*128 + Inz;

	extern __shared__ float input[];

	int stride_768[6] = {0, 768, 1536, 2304, 3072, 3840}; // 768 = 6*128
	for (int i = 0; i < 6; i++) {
		input[c_input + stride_768[i]] = pInputs[c_glb_start + i*stride_r];
	}
	__syncthreads();

	float BTd[6];
	switch(Iny1) {
		case 0:
			for (int j = 0; j < 6; j++) {
				BTd[j] = d(input, 0, j, Inz)*4 - d(input, 2, j, Inz)*5 + d(input, 4, j, Inz);
			}
			break;
		case 1:
			for (int j = 0; j < 6; j++) {
				BTd[j] = -d(input, 1, j, Inz)*4 - d(input, 2, j, Inz)*4 + d(input, 3, j, Inz) + d(input, 4, j, Inz);
			}
			break;
		case 2:
			for (int j = 0; j < 6; j++) {
				BTd[j] = d(input, 1, j, Inz)*4 - d(input, 2, j, Inz)*4 - d(input, 3, j, Inz) + d(input, 4, j, Inz);
			}
			break;
		case 3:
			for (int j = 0; j < 6; j++) {
				BTd[j] = -d(input, 1, j, Inz)*2 - d(input, 2, j, Inz) + d(input, 3, j, Inz)*2 + d(input, 4, j, Inz);
			}
			break;
		case 4:
			for (int j = 0; j < 6; j++) {
				BTd[j] = d(input, 1, j, Inz)*2 - d(input, 2, j, Inz) - d(input, 3, j, Inz)*2 + d(input, 4, j, Inz);
			}
			break;
		case 5:
			for (int j = 0; j < 6; j++) {
				BTd[j] = d(input, 1, j, Inz)*4 - d(input, 3, j, Inz)*5 + d(input, 5, j, Inz);
			}
			break;
	}
	__syncthreads();

	int tmp_offset = Iny1*768+Inz;
	for (int i = 0; i < 6; i++) {
		input[tmp_offset + i*128] = BTd[i];
	}
	__syncthreads();

	float BTdB[6];
	switch(Iny1) {
		case 0:
			for (int i = 0; i < 6; i++) {
				BTdB[i] = 4*d(input, i, 0, Inz) - 5*d(input, i, 2, Inz) + d(input, i, 4, Inz);
			}
			break;
		case 1:
			for (int i = 0; i < 6; i++) {
				BTdB[i] = -4*d(input, i, 1, Inz) - 4*d(input, i, 2, Inz) + d(input, i, 3, Inz) + d(input, i, 4, Inz);
			}
			break;
		case 2:
			for (int i = 0; i < 6; i++) {
				BTdB[i] = 4*d(input, i, 1, Inz) - 4*d(input, i, 2, Inz) - d(input, i, 3, Inz) + d(input, i, 4, Inz);
			}
			break;
		case 3:
			for (int i = 0; i < 6; i++) {
				BTdB[i] = -2*d(input, i, 1, Inz) - d(input, i, 2, Inz) + 2*d(input, i, 3, Inz) + d(input, i, 4, Inz);
			}
			break;
		case 4:
			for (int i = 0; i < 6; i++) {
				BTdB[i] = 2*d(input, i, 1, Inz) - d(input, i, 2, Inz) - 2*d(input, i, 3, Inz) + d(input, i, 4, Inz);
			}
			break;
		case 5:
			for (int i = 0; i < 6; i++) {
				BTdB[i] = 4*d(input, i, 1, Inz) - 5*d(input, i, 3, Inz) + d(input, i, 5, Inz);
			}
			break;
	}
	__syncthreads();

	for (int i = 0; i < 6; i++) {
		pOutputs[(Iny1 + i*6)*4096 + (blockIdx.x*4+blockIdx.y)*256 + Inz + (Part<<7)] = BTdB[i];
	}
}

__global__ void kernel_256_winograd_AtIA(float *pInputs, float *pBiases, float *pScales, float *pOutputs) {
	int Tilex = blockIdx.x, Tiley = blockIdx.y, Iny = threadIdx.y, kz = blockIdx.z, Inx = threadIdx.x;
	int c_input = Inx*6 + Iny;

	__shared__ float bias, scale;
	extern __shared__ float input[];

	input[c_input] = pInputs[c_input*16*256 + (Tilex*4+Tiley)*256 + kz];
	bias = pBiases[kz];
	scale = pScales[kz];
	__syncthreads();

	float tmp = 0;
	switch(Inx) {
		case 0:
			tmp = input[Iny] + input[6+Iny] + input[12+Iny] + input[18+Iny] + input[24+Iny];
			break;
		case 1:
			tmp = input[6+Iny] - input[12+Iny] + 2*input[18+Iny] - 2*input[24+Iny];
			break;
		case 2:
			tmp = input[6+Iny] + input[12+Iny] + 4*input[18+Iny] + 4*input[24+Iny];
			break;
		case 3:
			tmp = input[6+Iny] - input[12+Iny] + 8*input[18+Iny] - 8*input[24+Iny] + input[30+Iny];
			break;
	}
	__syncthreads();

	input[c_input] = tmp;
	__syncthreads();

	if (Inx > 3 || (Tilex == 3 && Inx > 1)) return;
	
	int x;
	float o;
	switch(Iny) {
		case 0:
			x = Inx*6;
			o = scale*(input[x]+input[x+1]+input[x+2]+input[x+3]+input[x+4]) + bias;
			pOutputs[(((Tilex<<2)+1+Inx)*16 + (Tiley<<2)+1)*256 + kz] = o > 0 ? o : 0;
			break;
		case 1:
			x = Inx*6;
			o = scale*(input[x+1] - input[x+2] + 2*input[x+3] - 2*input[x+4]) + bias;
			pOutputs[(((Tilex<<2)+1+Inx)*16 + (Tiley<<2)+2)*256 + kz] = o > 0 ? o : 0;
			break;
		case 2:
			if (Tiley == 3) break;
			x = Inx*6;
			o = scale*(input[x+1] + input[x+2] + 4*input[x+3] + 4*input[x+4]) + bias;
			pOutputs[(((Tilex<<2)+1+Inx)*16 + (Tiley<<2)+3)*256 + kz] = o > 0 ? o : 0;
			break;
		case 3:
			if (Tiley == 3) break;
			x = Inx*6;
			o = scale*(input[x+1] - input[x+2] + 8*input[x+3] - 8*input[x+4] + input[x+5]) + bias;
			pOutputs[(((Tilex<<2)+1+Inx)*16 + (Tiley<<2)+4)*256 + kz] = o > 0 ? o : 0;
			break;
	}
}

__global__ void kernel_256_OuterProduct_256(float *A, float *B, float *C) {
	int Tile = blockIdx.x, Part = blockIdx.y, tX = threadIdx.x, tY = threadIdx.y;
	int c_input = tY*256 + tX, c_kernel = c_input, T_offset = (Tile<<12) + (Part<<11) + c_input, B_offset = (Tile<<16) + c_kernel;
	
	extern __shared__ float input[];
	float *kernel = input + 2048, *out = kernel + 8192;
	int B_stride[32] = {0, 256, 512, 768, 1024, 1280, 1536, 1792, 2048, 2304, 2560, 2816, 3072, 3328, 3584, 3840, 4096, 4352, 4608, 4864, 5120, 5376, 5632, 5888, 6144, 6400, 6656, 6912, 7168, 7424, 7680, 7936};
	out[c_input] = 0.0f;
	out[c_input+1024] = 0;

	input[c_input] = A[T_offset];
	input[c_input+1024] = A[T_offset+1024];

	for (int k = 0; k < 8; k++) {
		int B_start = B_offset + (k<<13); // 32*64
		kernel[c_kernel] = B[B_start], kernel[c_kernel+1024] = B[B_start+1024];
		kernel[c_kernel+2048] = B[B_start+2048], kernel[c_kernel+3072] = B[B_start+3072];
		kernel[c_kernel+4096] = B[B_start+4096], kernel[c_kernel+5120] = B[B_start+5120];
		kernel[c_kernel+6144] = B[B_start+6144], kernel[c_kernel+7168] = B[B_start+7168];

		__syncthreads();

		float sum = 0, sum1 = 0;
		int y_tmp = (tY<<8)+(k<<5), y_tmp1 = y_tmp+1024;
		for (int j = 0; j < 32; j++) {
			sum += input[y_tmp + j] * kernel[tX + B_stride[j]];
			sum1 += input[y_tmp1 + j] * kernel[tX + B_stride[j]];
		}
		out[c_input] += sum;
		out[c_input+1024] += sum1;
		__syncthreads();
	}

	C[T_offset] = out[c_input];
	C[T_offset+1024] = out[c_input+1024];
}

int kernel_256() {
	float *input_ = get_parameter(inputName256, 16*16*256);
	float *bias = get_parameter(biasName256, 256);
	float *input, *output, *l_weights, *l_bias;
	uint64_t nT1 = 0, nT2 = 0, nT1_cudnn = 0, nT2_cudnn = 0;
	hipError_t s;



	/////////////////////////////////

	// My Kernel

	/////////////////////////////////
	float *kernel = get_parameter(weight_winograd_Name256, 36*256*256), *t_input, *ip;
	int nInput = 16*16*256, nOutput = 16*16*256, nWeights = 36*256*256, nBias = 256, nTransInput = 16*6*6*256, nInnerProd = 16*6*6*256;
	float *l_bnBias, *l_bnScale, *bnBias, *bnScale;

	hipMalloc((void **) &input, nInput<<3);
	hipMalloc((void **) &output, nOutput<<2);
	hipMalloc((void **) &l_weights, nWeights<<2);
	hipMalloc((void **) &l_bias, nBias<<2);
	hipMalloc((void **) &t_input, nTransInput<<2);
	hipMalloc((void **) &ip, nInnerProd<<2);

	hipMemset((void *) input, 0, nInput<<3);
	hipMemset((void *) output, 0, nOutput<<2);
	hipMemset((void *) t_input, 0, nTransInput<<2);
	hipMemset((void *) l_weights, 0, nWeights<<2);
	hipMemset((void *) ip, 0, nInnerProd<<2);

	hipMemcpy(input, input_, nInput<<2, hipMemcpyHostToDevice);
	hipMemcpy(l_weights, kernel, nWeights<<2, hipMemcpyHostToDevice);
	hipMemcpy(l_bias, bias, nBias<<2, hipMemcpyHostToDevice);

	bnBias = get_parameter(bnBias_winograd_Name256, 256);
	bnScale = get_parameter(bnScale_winograd_Name256, 256);
	hipMalloc((void **) &l_bnBias, nBias<<2);
	hipMalloc((void **) &l_bnScale, nBias<<2);
	hipMemcpy(l_bnBias, bnBias, nBias<<2, hipMemcpyHostToDevice);
	hipMemcpy(l_bnScale, bnScale, nBias<<2, hipMemcpyHostToDevice);

	float tmp[nOutput];

	nT1 = getTimeMicroseconds64();

	kernel_256_winograd_BtdB <<<dim3(4, 4, 2), dim3(128, 6), (6*6*128)<<2 >>> (input, t_input);
	kernel_256_OuterProduct_256<<<dim3(36, 2), dim3(256, 4), (8*256 + 32*256 + 8*256)<<2 >>> (t_input, l_weights, ip);
	kernel_256_winograd_AtIA <<<dim3(4, 4, 256), dim3(6, 6), ((6*6)<<2)>>> (ip, l_bnBias, l_bnScale, output);
	//cudaCheckError();
	hipDeviceSynchronize();
	
	nT2 = getTimeMicroseconds64();
	printf("TotalTime = %d us\n", nT2-nT1); 

	s = hipMemcpy(tmp, output, nOutput<<2, hipMemcpyDeviceToHost);
	printf("%s\n", hipGetErrorName(s));
	//cudaCheckError();

	hipFree(t_input);
	hipFree(output);
	hipFree(l_weights);
	hipFree(l_bias);
	hipFree(ip);

	free(kernel);
	free(bnScale);
	free(bnBias);



	/////////////////////////////////

	// cuDNN

	/////////////////////////////////
	kernel = get_parameter(weight_NCHW_Name256, 9*256*256);
	bnBias = get_parameter(bnBiasName256, 256);
	bnScale = get_parameter(bnScaleName256, 256);
	float* eMean = get_parameter(eMeanName256, 256);
	float* eVar = get_parameter(eVarName256, 256);
	float *l_eMean, *l_eVar;
	nInput = 16*16*256, nOutput = 14*14*256, nWeights = 3*3*256*256, nBias = 256;

	hipMalloc((void **) &output, nOutput<<2);
	hipMalloc((void **) &l_weights, nWeights<<2);
	hipMalloc((void **) &l_bias, nBias<<2);
	hipMemcpy(l_weights, kernel, nWeights<<2, hipMemcpyHostToDevice);
	hipMemcpy(l_bias, bias, nBias<<2, hipMemcpyHostToDevice);

	hipMalloc((void **) &l_eMean, nBias<<2);
	hipMalloc((void **) &l_eVar, nBias<<2);
	hipMemcpy(l_bnBias, bnBias, nBias<<2, hipMemcpyHostToDevice);
	hipMemcpy(l_bnScale, bnScale, nBias<<2, hipMemcpyHostToDevice);
	hipMemcpy(l_eMean, eMean, nBias<<2, hipMemcpyHostToDevice);
	hipMemcpy(l_eVar, eVar, nBias<<2, hipMemcpyHostToDevice);

	hipMemset((void *) output, 0, nOutput<<2);

	float tmp_cudnn[nOutput];

	hipdnnStatus_t status;
	float one = 1.0, zero = 0.0;
	int size;

	hipdnnHandle_t handle;
	status = hipdnnCreate(&handle);
	if (status != HIPDNN_STATUS_SUCCESS) printf("failed1\n");

	hipdnnTensorDescriptor_t xdesc, ydesc, bdesc;
	hipdnnFilterDescriptor_t wdesc; // HIPDNN_TENSOR_NHWC, HIPDNN_TENSOR_NCHW
	status = hipdnnCreateTensorDescriptor(&xdesc);
	if (status != HIPDNN_STATUS_SUCCESS) printf("failed2\n");
	status = hipdnnSetTensor4dDescriptor(xdesc, HIPDNN_TENSOR_NHWC, HIPDNN_DATA_FLOAT, 1, 256, 16, 16);
	if (status != HIPDNN_STATUS_SUCCESS) printf("failed3\n");
	status = hipdnnCreateTensorDescriptor(&ydesc);
	if (status != HIPDNN_STATUS_SUCCESS) printf("failed4\n");
	status = hipdnnSetTensor4dDescriptor(ydesc, HIPDNN_TENSOR_NHWC, HIPDNN_DATA_FLOAT, 1, 256, 14, 14);
	if (status != HIPDNN_STATUS_SUCCESS) printf("failed5\n");
	status = hipdnnCreateFilterDescriptor(&wdesc);
	if (status != HIPDNN_STATUS_SUCCESS) printf("failed6\n");
	status = hipdnnSetFilter4dDescriptor(wdesc, HIPDNN_DATA_FLOAT, HIPDNN_TENSOR_NCHW, 256, 256, 3, 3);
	if (status != HIPDNN_STATUS_SUCCESS) printf("failed7\n");
	status = hipdnnCreateTensorDescriptor(&bdesc);
	if (status != HIPDNN_STATUS_SUCCESS) printf("failed8\n");
	status = hipdnnSetTensor4dDescriptor(bdesc, HIPDNN_TENSOR_NHWC, HIPDNN_DATA_FLOAT, 1, 256, 1, 1);
	if (status != HIPDNN_STATUS_SUCCESS) printf("failed9\n");
	hipdnnConvolutionDescriptor_t conv_desc;
	status = hipdnnCreateConvolutionDescriptor(&conv_desc);
	if (status != HIPDNN_STATUS_SUCCESS) printf("failed10\n");
	status = hipdnnSetConvolution2dDescriptor(conv_desc, 0,0, 1,1,1,1, HIPDNN_CROSS_CORRELATION, HIPDNN_DATA_FLOAT); //HIPDNN_CONVOLUTION
	if (status != HIPDNN_STATUS_SUCCESS) printf("failed11\n");

	hipdnnActivationDescriptor_t act_desc;
	status = hipdnnCreateActivationDescriptor(&act_desc);  
	if (status != HIPDNN_STATUS_SUCCESS) printf("failed12\n");
	status = hipdnnSetActivationDescriptor(act_desc, HIPDNN_ACTIVATION_RELU, HIPDNN_NOT_PROPAGATE_NAN, 0);
	if (status != HIPDNN_STATUS_SUCCESS) printf("failed13\n");

	hipdnnTensorDescriptor_t bnScaleBiasMeanVarDesc;
	status = hipdnnCreateTensorDescriptor(&bnScaleBiasMeanVarDesc);
	if (status != HIPDNN_STATUS_SUCCESS) printf("failed14\n");
	status = hipdnnSetTensor4dDescriptor(bnScaleBiasMeanVarDesc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, 1, 256, 1, 1);
	if (status != HIPDNN_STATUS_SUCCESS) printf("failed15\n");

	hipdnnConvolutionFwdAlgo_t algo = (hipdnnConvolutionFwdAlgo_t)6;

	status = hipdnnGetConvolutionForwardWorkspaceSize(handle,
	   xdesc,
	   wdesc,
	   conv_desc,
	   ydesc,
	   algo,
	   (size_t *)&(size));

	float *extra;
	hipMalloc((void **) &extra, size);
	
	nT1_cudnn = getTimeMicroseconds64();

	status = hipdnnConvolutionForward(handle, &one,
		xdesc, input, wdesc, l_weights, 
		conv_desc, algo, 
		extra, size, &zero,
		ydesc, output);
	if (status != HIPDNN_STATUS_SUCCESS) printf("Not Successed1\n");

	status = hipdnnBatchNormalizationForwardInference(handle, HIPDNN_BATCHNORM_SPATIAL,
		&one, &zero, 
		ydesc, output, ydesc, output,
		bnScaleBiasMeanVarDesc, l_bnScale, l_bnBias, l_eMean, l_eVar, HIPDNN_BN_MIN_EPSILON);
	if (status != HIPDNN_STATUS_SUCCESS) printf("Not Successed2\n");

	status = hipdnnActivationForward(handle, act_desc, &one,
		ydesc, output, &zero,
		ydesc, output);
	if (status != HIPDNN_STATUS_SUCCESS) printf("Not Successed3\n");

	hipDeviceSynchronize();
	nT2_cudnn = getTimeMicroseconds64();
	printf("cuDNN TotalTime = %d us\n", nT2_cudnn-nT1_cudnn);
	
	s = hipMemcpy(tmp_cudnn, output, nOutput<<2, hipMemcpyDeviceToHost);
	printf("%s\n", hipGetErrorName(s));


	hipFree(extra);
	hipFree(input);
	hipFree(output);
	hipFree(l_weights);
	hipFree(l_bias);

	hipFree(l_bnScale);
	hipFree(l_bnBias);
	hipFree(l_eMean);
	hipFree(l_eVar);

	free(bias);
	free(kernel);

	free(bnScale);
	free(bnBias);
	free(eMean);
	free(eVar);
	free(input_);

	output_checker(tmp, tmp_cudnn, 14, 256, 1);

	return ((nT2-nT1) << 16) | (nT2_cudnn-nT1_cudnn);
}